
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 512
#define BLOCK_MASK (BLOCK_SIZE)*2
__global__ void avg_kernel(float* in_vec, float* out_vec, int len)
{
    __shared__ float shared_seg_sum[2 * BLOCK_SIZE];
    unsigned int p = 0;
    do {
	if (len > blockIdx.x * BLOCK_MASK + threadIdx.x + p * BLOCK_SIZE)
	    shared_seg_sum[threadIdx.x + p * BLOCK_SIZE] = in_vec[blockIdx.x * BLOCK_MASK + threadIdx.x + p * BLOCK_SIZE];
	else
	    shared_seg_sum[threadIdx.x + p * BLOCK_SIZE] = 0.0f;
	p++;
    } while (p <= 1);
    
    for (unsigned int m = BLOCK_SIZE; m >= 1; m /= 2) {
	if (m > threadIdx.x) {
	    shared_seg_sum[threadIdx.x] += shared_seg_sum[threadIdx.x + m];
	}
	__syncthreads();
    }
    (threadIdx.x == 0) ? out_vec[blockIdx.x] = shared_seg_sum[threadIdx.x] / (1.0*len) : 0.0f;
}
